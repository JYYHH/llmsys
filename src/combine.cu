#include <hip/hip_runtime.h>
#include <assert.h>
#include <iostream>
#include <sstream>
#include <fstream>

#define MAX_DIMS 10
#define TILE 32
#define BASE_THREAD_NUM 32

#define ADD_FUNC       1
#define MUL_FUNC       2
#define ID_FUNC        3
#define NEG_FUNC       4
#define LT_FUNC        5
#define EQ_FUNC        6
#define SIGMOID_FUNC   7
#define RELU_FUNC      8
#define RELU_BACK_FUNC 9
#define LOG_FUNC       10
#define LOG_BACK_FUNC  11
#define EXP_FUNC       12
#define INV_FUNC       13
#define INV_BACK_FUNC  14
#define IS_CLOSE_FUNC  15
#define MAX_FUNC       16
#define POW            17
#define TANH           18

__device__ float fn(int fn_id, float x, float y=0) {
    switch(fn_id) {
      case ADD_FUNC: {
        return x + y;
      }
      case MUL_FUNC: {
        return x * y;
      }
      case ID_FUNC: {
      	return x;
      }
      case NEG_FUNC: {
        return -x;
      }
      case LT_FUNC: {
        if (x < y) {
          return 1.0;
        }
        else {
          return 0.0;
        }
      }
      case EQ_FUNC: {
        if (x == y) {
          return 1.0;
        }
        else {
          return 0.0;
        }
      }
      case SIGMOID_FUNC: {
        if (x >= 0) {
          return 1.0 / (1.0 + exp(-x));
        }
        else {
          return exp(x) / (1.0 + exp(x));
        }
      }
      case RELU_FUNC: {
        return max(x, 0.0);
      }
      case RELU_BACK_FUNC: {
        if (x > 0) {
          return y;
        }
        else {
          return 0.0;
        }
      }
      case LOG_FUNC: {
        return log(x + 1e-6);
      }
      case LOG_BACK_FUNC: {
        return y / (x + 1e-6);
      }
      case EXP_FUNC: {
        return exp(x);
      }
      case INV_FUNC: {
        return float(1.0 / x);
      }
      case INV_BACK_FUNC: {
        return -(1.0 / (x * x)) * y;
      }
      case IS_CLOSE_FUNC: {
        return (x - y < 1e-2) && (y - x < 1e-2);
      }
      case MAX_FUNC: {
        if (x > y) {
          return x;
        }
        else {
          return y;
        }
      }
      case POW: {
        return pow(x, y);
      }
      case TANH: {
        return tanh(x);
      }
      default: {
        return x + y;
      }
    }
    
}


__device__ int index_to_position(const int* index, const int* strides, int num_dims) {
    int position = 0;
    for (int i = 0; i < num_dims; ++i) {
        position += index[i] * strides[i];
    }
    return position;
}

__device__ void to_index(int ordinal, const int* shape, int* out_index, int num_dims) {
    int cur_ord = ordinal;
    for (int i = num_dims - 1; i >= 0; --i) {
        int sh = shape[i];
        out_index[i] = cur_ord % sh;
        cur_ord /= sh;
    }
}

__device__ void broadcast_index(const int* big_index, const int* big_shape, const int* shape, int* out_index, int num_dims_big, int num_dims) {
    for (int i = 0; i < num_dims; ++i) {
        if (shape[i] > 1) {
            out_index[i] = big_index[i + (num_dims_big - num_dims)];
        } else {
            out_index[i] = 0;
        }
    }
}


__global__ void MatrixMultiplyKernel(
    float* out,
    const int* out_shape,
    const int* out_strides,
    float* a_storage,
    const int* a_shape,
    const int* a_strides,
    float* b_storage,
    const int* b_shape,
    const int* b_strides
) {

  __shared__ float a_shared[TILE][TILE];
  __shared__ float b_shared[TILE][TILE];

  // In each block, we will compute a batch of the output matrix
  // All the threads in the block will work together to compute this batch
  int batch = blockIdx.z;
  int a_batch_stride = a_shape[0] > 1 ? a_strides[0] : 0;
  int b_batch_stride = b_shape[0] > 1 ? b_strides[0] : 0;
  int out_batch_stride = out_shape[0] > 1 ? out_strides[0] : 0;
  const int row_base = blockIdx.x * blockDim.x;
  const int col_base = blockIdx.y * blockDim.y;
  const int row_offset = threadIdx.x;
  const int col_offset = threadIdx.y;
  const int row = row_base + row_offset;
  const int col = col_base + col_offset;
  int col_a = col_offset;
  int row_b = row_offset;
  int reduce_dim = (a_shape[2] + TILE - 1) / TILE;
  float ret = 0.0; // one element in out matrix

  // walk through
  for (int i = 0; i < reduce_dim; i++, col_a += TILE, row_b += TILE){
    // read the block in a and b into the shared memory
    a_shared[row_offset][col_offset] = (row < a_shape[1] && col_a < a_shape[2]) ? a_storage[batch * a_batch_stride + row * a_strides[1] + col_a * a_strides[2]] : 0.0;
      // small trick here: we just save the transpose of the submatrix...
    b_shared[col_offset][row_offset] = (row_b < b_shape[1] && col < b_shape[2]) ? b_storage[batch * b_batch_stride + row_b * b_strides[1] + col * b_strides[2]] : 0.0;
    // sync all threads in this block
    __syncthreads();
    // then do the local reduction here
    for (int j = 0; j < TILE; j++)
      ret += a_shared[row_offset][j] * b_shared[col_offset][j];
    // must have a sync, since next we will load sth. into the shared memory
    __syncthreads();
  }

  // save the result
  if (row < out_shape[1] && col < out_shape[2])
    out[batch * out_batch_stride + row * out_strides[1]+ col * out_strides[2]] = ret;
}


__global__ void mapKernel(
    float* out, 
    int* out_shape, 
    int* out_strides, 
    int out_size, 
    float* in_storage, 
    int* in_shape, 
    int* in_strides,
    int shape_size,
    int fn_id
) {
  int out_index[MAX_DIMS];
  int in_index[MAX_DIMS];
  int position = blockIdx.x * blockDim.x + threadIdx.x;
  int in_position = 0;

  if (position >= out_size) // whether is out of bound
    return;
  // out_position to out index
  to_index(position, out_shape, out_index, shape_size); 
  // out index to in index, maybe need broadcast
  broadcast_index(out_index, out_shape, in_shape, in_index, shape_size, shape_size);
  // in index to in_position
  in_position = index_to_position(in_index, in_strides, shape_size);
  out[position] = fn(fn_id, in_storage[in_position]);
}


__global__ void reduceKernel(
    float* out, 
    int* out_shape, 
    int* out_strides, 
    int out_size, 
    float* a_storage, 
    int* a_shape, 
    int* a_strides, 
    int reduce_dim,
    float reduce_value,
    int shape_size,
    int fn_id
) {
  int out_index[MAX_DIMS];
  int position = blockIdx.x * blockDim.x + threadIdx.x;
  float result = reduce_value;
  int reduce_dim_l = a_shape[reduce_dim];

  if (position >= out_size) // whether is out of bound
    return;
  // out_position to out index
  to_index(position, out_shape, out_index, shape_size);
  // walk through
  for (int i = 0; i < reduce_dim_l; i++){
    out_index[reduce_dim] = i;
    result = fn(fn_id, result, a_storage[index_to_position(out_index, a_strides, shape_size)]);
  }
  // save output
  out[position] = result;
}

__global__ void zipKernel(
    float* out, 
    int* out_shape, 
    int* out_strides, 
    int out_size,
    int out_shape_size,
    float* a_storage, 
    int* a_shape, 
    int* a_strides,
    int a_shape_size,
    float* b_storage, 
    int* b_shape, 
    int* b_strides,
    int b_shape_size,
    int fn_id
) {
  int out_index[MAX_DIMS];
  int a_index[MAX_DIMS];
  int b_index[MAX_DIMS];
  int position = blockIdx.x * blockDim.x + threadIdx.x;
  int a_position = 0, b_position = 0;

  if (position >= out_size) // whether is out of bound
    return;
  // out_position to out index
  to_index(position, out_shape, out_index, out_shape_size); 
  // out index to in indices, maybe need broadcast
  broadcast_index(out_index, out_shape, a_shape, a_index, out_shape_size, a_shape_size);
  broadcast_index(out_index, out_shape, b_shape, b_index, out_shape_size, b_shape_size);
  // in indices to in_position
  a_position = index_to_position(a_index, a_strides, a_shape_size);
  b_position = index_to_position(b_index, b_strides, b_shape_size);
  // compute the operation
  out[position] = fn(fn_id, a_storage[a_position], b_storage[b_position]);
}


extern "C" {

void MatrixMultiply(
    float* out,
    int* out_shape,
    int* out_strides,
    float* a_storage,
    int* a_shape,
    int* a_strides,
    float* b_storage,
    int* b_shape,
    int* b_strides,
    int batch, int m, int p
) {
    int n = a_shape[2];

    // Allocate device memory
    float *d_out, *d_a, *d_b;
    hipMalloc(&d_a, batch * m * n * sizeof(float));
    hipMalloc(&d_b, batch * n * p * sizeof(float));
    hipMalloc(&d_out, batch * m * p * sizeof(float));

    int *d_out_shape, *d_out_strides, *d_a_shape, *d_a_strides, *d_b_shape, *d_b_strides;
    hipMalloc(&d_out_shape, 3 * sizeof(int));
    hipMalloc(&d_out_strides, 3 * sizeof(int));
    hipMalloc(&d_a_shape, 3 * sizeof(int));
    hipMalloc(&d_a_strides, 3 * sizeof(int));
    hipMalloc(&d_b_shape, 3 * sizeof(int));
    hipMalloc(&d_b_strides, 3 * sizeof(int));


    // Copy data to the device
    hipMemcpy(d_a, a_storage, batch * m * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b_storage, batch * n * p * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_out_shape, out_shape, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_out_strides, out_strides, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_shape, a_shape, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_strides, a_strides, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b_shape, b_shape, 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b_strides, b_strides, 3 * sizeof(int), hipMemcpyHostToDevice);

    int threadsPerBlock = BASE_THREAD_NUM;
    dim3 blockDims(threadsPerBlock, threadsPerBlock, 1); // Adjust these values based on your specific requirements
    dim3 gridDims((m + threadsPerBlock - 1) / threadsPerBlock, (p + threadsPerBlock - 1) / threadsPerBlock, batch);
    MatrixMultiplyKernel<<<gridDims, blockDims>>>(
        d_out, d_out_shape, d_out_strides, d_a, d_a_shape, d_a_strides, d_b, d_b_shape, d_b_strides
    );

    // Copy back to the host
    hipMemcpy(out, d_out, batch * m * p * sizeof(float), hipMemcpyDeviceToHost);
    
    hipDeviceSynchronize();

    // Check CUDA execution
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Matmul Error: %s\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

    // Free memory on device
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
    hipFree(d_out_shape);
    hipFree(d_out_strides);
    hipFree(d_a_shape);
    hipFree(d_a_strides);
    hipFree(d_b_shape);
    hipFree(d_b_strides);
}

void tensorMap(
    float* out, 
    int* out_shape, 
    int* out_strides, 
    int out_size, 
    float* in_storage, 
    int* in_shape, 
    int* in_strides,
    int in_size,
    int shape_size,
    int fn_id
) {

    float *d_out, *d_in;
    hipMalloc(&d_out, out_size * sizeof(float));
    hipMalloc(&d_in, in_size * sizeof(float));

    int *d_out_shape, *d_out_strides, *d_in_shape, *d_in_strides;
    hipMalloc(&d_out_shape, shape_size * sizeof(int));
    hipMalloc(&d_out_strides, shape_size * sizeof(int));
    hipMalloc(&d_in_shape, shape_size * sizeof(int));
    hipMalloc(&d_in_strides, shape_size * sizeof(int));

    hipMemcpy(d_in, in_storage, in_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_out_shape, out_shape, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_out_strides, out_strides, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_in_shape, in_shape, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_in_strides, in_strides, shape_size * sizeof(int), hipMemcpyHostToDevice);
    
    int threadsPerBlock = BASE_THREAD_NUM;
    int blocksPerGrid = (out_size + threadsPerBlock - 1) / threadsPerBlock;
    mapKernel<<<blocksPerGrid, threadsPerBlock>>>(
      d_out, d_out_shape, d_out_strides, out_size, 
      d_in, d_in_shape, d_in_strides, 
      shape_size, fn_id);
    
    // Copy back to the host
    hipMemcpy(out, d_out, out_size * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    // Check CUDA execution
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Map Error: %s\n", hipGetErrorString(err));
      // Handle the error (e.g., by exiting the program)
      exit(EXIT_FAILURE);
    }

    // Free memory on device
    hipFree(d_in);
    hipFree(d_out);
    hipFree(d_out_shape);
    hipFree(d_out_strides);
    hipFree(d_in_shape);
    hipFree(d_in_strides);
}


void tensorZip(
    float* out, 
    int* out_shape, 
    int* out_strides, 
    int out_size,
    int out_shape_size,
    float* a_storage, 
    int* a_shape, 
    int* a_strides,
    int a_size,
    int a_shape_size,
    float* b_storage, 
    int* b_shape, 
    int* b_strides,
    int b_size,
    int b_shape_size,
    int fn_id
) {

    // Allocate device memory
    float *d_out, *d_a, *d_b;
    hipMalloc((void **)&d_a, a_size * sizeof(float));
    hipMalloc(&d_b, b_size * sizeof(float));
    hipMalloc(&d_out, out_size * sizeof(float));

    int *d_out_shape, *d_out_strides, *d_a_shape, *d_a_strides, *d_b_shape, *d_b_strides;
    hipMalloc(&d_out_shape, out_shape_size * sizeof(int));
    hipMalloc(&d_out_strides, out_shape_size * sizeof(int));
    hipMalloc(&d_a_shape, a_shape_size * sizeof(int));
    hipMalloc(&d_a_strides, a_shape_size * sizeof(int));
    hipMalloc(&d_b_shape, b_shape_size * sizeof(int));
    hipMalloc(&d_b_strides, b_shape_size * sizeof(int));

    // Copy data to the device
    hipMemcpy(d_a, a_storage, a_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b_storage, b_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_out_shape, out_shape, out_shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_out_strides, out_strides, out_shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_shape, a_shape, a_shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_strides, a_strides, a_shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b_shape, b_shape, b_shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b_strides, b_strides, b_shape_size * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel
    int threadsPerBlock = BASE_THREAD_NUM;
    int blocksPerGrid = (out_size + threadsPerBlock - 1) / threadsPerBlock;
    zipKernel<<<blocksPerGrid, threadsPerBlock>>>(
      d_out, d_out_shape, d_out_strides, out_size, out_shape_size,
      d_a, d_a_shape, d_a_strides, a_shape_size,
      d_b, d_b_shape, d_b_strides, b_shape_size,
      fn_id);

    // Copy back to the host
    hipMemcpy(out, d_out, out_size * sizeof(float), hipMemcpyDeviceToHost);
    
    hipDeviceSynchronize();


    // Check CUDA execution
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Zip Error: %s\n", hipGetErrorString(err));
      // Handle the error (e.g., by exiting the program)
      exit(EXIT_FAILURE);
    }

    // Free memory on device
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
    hipFree(d_out_shape);
    hipFree(d_out_strides);
    hipFree(d_a_shape);
    hipFree(d_a_strides);
    hipFree(d_b_shape);
    hipFree(d_b_strides);
}



void tensorReduce(
    float* out, 
    int* out_shape, 
    int* out_strides, 
    int out_size, 
    float* a_storage, 
    int* a_shape, 
    int* a_strides, 
    int reduce_dim, 
    float reduce_value,
    int shape_size,
    int fn_id
) {
    int a_size = out_size * a_shape[reduce_dim];
    float *d_out, *d_a;
    hipMalloc(&d_out, out_size * sizeof(float));
    hipMalloc(&d_a, a_size * sizeof(float));

    int *d_out_shape, *d_out_strides, *d_a_shape, *d_a_strides;
    hipMalloc(&d_out_shape, shape_size * sizeof(int));
    hipMalloc(&d_out_strides, shape_size * sizeof(int));
    hipMalloc(&d_a_shape, shape_size * sizeof(int));
    hipMalloc(&d_a_strides, shape_size * sizeof(int));

    hipMemcpy(d_a, a_storage, a_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_out_shape, out_shape, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_out_strides, out_strides, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_shape, a_shape, shape_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a_strides, a_strides, shape_size * sizeof(int), hipMemcpyHostToDevice);
    
    int threadsPerBlock = BASE_THREAD_NUM;
    int blocksPerGrid = (out_size + threadsPerBlock - 1) / threadsPerBlock;
    reduceKernel<<<blocksPerGrid, threadsPerBlock>>>(
        d_out, d_out_shape, d_out_strides, out_size, 
        d_a, d_a_shape, d_a_strides, 
        reduce_dim, reduce_value, shape_size, fn_id
    );

    // Copy back to the host
    hipMemcpy(out, d_out, out_size * sizeof(float), hipMemcpyDeviceToHost);
    
    hipDeviceSynchronize();

    // Check CUDA execution
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Reduce Error: %s\n", hipGetErrorString(err));
      // Handle the error (e.g., by exiting the program)
      exit(EXIT_FAILURE);
    }

    hipFree(d_a);
    hipFree(d_out);
    hipFree(d_out_shape);
    hipFree(d_out_strides);
    hipFree(d_a_shape);
    hipFree(d_a_strides);
}

}